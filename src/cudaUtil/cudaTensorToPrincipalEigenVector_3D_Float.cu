#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <eigenUtil.h>
#include <cudaTensorToPrincipalEigenVector_3D_Float.h>


__global__ void cudaTensorToPrincipalEigenVector_3D_Float_Kernel(float* Txx, float* Txy, float* Txz, float* Tyy, float* Tyz, float* Tzz,
	long size, float* nx, float *ny, float* nz)
{
	const long add = blockIdx.x * blockDim.x + threadIdx.x;

	if (add < size )
	{
		EIGEN_PRINCIPAL_VECTOR_SDP3X3_F(Txx[add], Tyy[add], Tzz[add], Txy[add], Txz[add], Tyz[add], nx[add], ny[add], nz[add])
	}

}



void cudaTensorToPrincipalEigenVector_3D_Float(float* d_Txx, float* d_Txy, float* d_Txz, float* d_Tyy, float* d_Tyz, float* d_Tzz,
	long size, float* d_nx, float *d_ny, float* d_nz)
{
	dim3 block(1024);
	dim3 grid((size - 1) / block.x + 1);
	cudaTensorToPrincipalEigenVector_3D_Float_Kernel << <grid, block >> > (d_Txx, d_Txy, d_Txz, d_Tyy, d_Tyz, d_Tzz, size, d_nx, d_ny, d_nz);
	hipDeviceSynchronize();
}