#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudaConvolution_2Inputs_3D_Float_Valid_Kernel.h>

// ========================================================
// KERNEL
// ========================================================

__global__ void cudaConvolution_2Inputs_3D_Float_Valid_X_Kernel(float* in1, float *in2, int dimx, int dimy, int dimz, float* mask, int size2, float* out)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x,
		y = blockIdx.y * blockDim.y + threadIdx.y,
		z = blockIdx.z * blockDim.z + threadIdx.z;

	// in += baseZ * (dimx * dimy) + baseY * dimx + baseX;
	// out += baseZ * ((dimx - 2 * size2) * dimy) + baseY * (dimx - 2 * size2) + baseX - size2;

	if (x >= size2 && x < dimx - size2 && y < dimy && z < dimz)
	{
		float sum = 0.0f;
#pragma unroll
		for (int i = -size2; i <= size2; i++)
			sum += mask[size2 - i] * ( in1[(long)dimx * dimy * z + (long)dimx * y + (long)x + i] * in2[(long)dimx * dimy * z + (long)dimx * y + (long)x + i]);
		out[(long)z * ((dimx - 2 * size2) * dimy) + (long)y * (dimx - 2 * size2) + (long)x - size2] = sum;
	}
}



__global__ void cudaConvolution_2Inputs_3D_Float_Valid_Y_Kernel(float* in1, float *in2, int dimx, int dimy, int dimz, float* mask, int size2, float* out)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x,
		y = blockIdx.y * blockDim.y + threadIdx.y,
		z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < dimx && y >= size2 && y < dimy - size2 && z < dimz)
	{
		float sum = 0.0f;
#pragma unroll
		for (int i = -size2; i <= size2; i++)
			sum += mask[size2 - i] * ( in1[dimx * dimy * z + dimx * (y + i) + x] * in2[dimx * dimy * z + dimx * (y + i) + x] );
		out[z * (dimx * (dimy - 2 * size2)) + (y - size2) * dimx + x] = sum;
	}
}


__global__ void cudaConvolution_2Inputs_3D_Float_Valid_Z_Kernel(float* in1, float *in2, int dimx, int dimy, int dimz, float* mask, int size2, float* out)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x,
		y = blockIdx.y * blockDim.y + threadIdx.y,
		z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < dimx && y < dimy && z >= size2 && z < dimz - size2)
	{
		float sum = 0.0f;
#pragma unroll
		for (int i = -size2; i <= size2; i++)
			sum += mask[size2 - i] * ( in1[dimx * dimy * (z + i) + dimx * y + x] * in2[dimx * dimy * (z + i) + dimx * y + x] );
		out[(z - size2) * dimx * dimy + y * dimx + x] = sum;
	}
}




void cudaConvolution_2Inputs_3D_Float_Valid_X(float* d_in1, float *d_in2, int dimx, int dimy, int dimz, float* d_mask, int maskSize, float* d_out)
{
	dim3 block(10, 10, 10);
	dim3 grid((dimx - 1) / block.x + 1, (dimy - 1) / block.y + 1, (dimz - 1) / block.z + 1);
	cudaConvolution_2Inputs_3D_Float_Valid_X_Kernel << <grid, block >> > (d_in1, d_in2, dimx, dimy, dimz, d_mask, maskSize / 2, d_out);
	hipDeviceSynchronize();
}

void cudaConvolution_2Inputs_3D_Float_Valid_Y(float* d_in1, float *d_in2, int dimx, int dimy, int dimz, float* d_mask, int maskSize, float* d_out)
{
	dim3 block(10, 10, 10);
	dim3 grid((dimx - 1) / block.x + 1, (dimy - 1) / block.y + 1, (dimz - 1) / block.z + 1);
	cudaConvolution_2Inputs_3D_Float_Valid_Y_Kernel << <grid, block >> > (d_in1, d_in2, dimx, dimy, dimz, d_mask, maskSize / 2, d_out);
	hipDeviceSynchronize();
}

void cudaConvolution_2Inputs_3D_Float_Valid_Z(float* d_in1, float *d_in2, int dimx, int dimy, int dimz, float* d_mask, int maskSize, float* d_out)
{
	dim3 block(10, 10, 10);
	dim3 grid((dimx - 1) / block.x + 1, (dimy - 1) / block.y + 1, (dimz - 1) / block.z + 1);
	cudaConvolution_2Inputs_3D_Float_Valid_Z_Kernel << <grid, block >> > (d_in1, d_in2, dimx, dimy, dimz, d_mask, maskSize / 2, d_out);
	hipDeviceSynchronize();
}

// #define CUDA_MEM_CPY_TO_SYMBOL_FLOAT(_dst, _src, _size) hipMemcpyToSymbol(HIP_SYMBOL(_dst), _src, _size*sizeof(float));




/*

__global__ void cudaConvolution3DValidXKernel(float* in, int dimx, int dimy, int dimz, float *mask, int size2, float* out)
{
	__shared__ float s_Data[BLOCKDIMZ][BLOCKDIMY][BLOCKDIMX * 3];
	const int baseX = (blockIdx.x - 1) * blockDim.x + threadIdx.x,
		baseY = blockIdx.y * blockDim.y + threadIdx.y,
		baseZ = blockIdx.z * blockDim.z + threadIdx.z;

	in += baseZ * (dimx * dimy) + baseY * dimx + baseX;
	out += baseZ * ((dimx - 2 * size2) * dimy) + baseY * (dimx - 2 * size2) + baseX - size2;

	if (baseX < dimx && baseY < dimy && baseZ < dimz)
	{
		s_Data[threadIdx.z][threadIdx.y][threadIdx.x + BLOCKDIMX] = (baseX + BLOCKDIMX < dimx) ? in[BLOCKDIMX] : 0.0f;
		s_Data[threadIdx.z][threadIdx.y][threadIdx.x] = (baseX > 0) ? in[0] : 0.0f;
		s_Data[threadIdx.z][threadIdx.y][threadIdx.x + 2 * BLOCKDIMX] = (dimx - baseX > 2 * BLOCKDIMX) ? in[2 * BLOCKDIMX] : 0.0f;
		__syncthreads();

		if (baseX + BLOCKDIMX >= size2 && baseX + BLOCKDIMX + size2 < dimx)
		{
			float sum = 0.0f;

#pragma unroll
			for (int i = -size2; i <= size2; i++)
				sum += mask[size2 - i] * s_Data[threadIdx.z][threadIdx.y][threadIdx.x + BLOCKDIMX + i];
			out[BLOCKDIMX] = sum;
		}
	}
}

*/